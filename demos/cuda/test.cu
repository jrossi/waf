#include "hip/hip_runtime.h"


#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "test.h"

#define N 624
#define M 397
#define MATRIX_A 0x9908b0dfUL
#define UPPER_MASK 0x80000000UL
#define LOWER_MASK 0x7fffffffUL

#define DIFFLEN 2688
#define REALDIFFLEN 81792
#define NUMCAND 3072
#define ORISIZ 81776

#define DEVLEN (DIFFLEN * NUMCAND)


__device__ void init_gengrk(unsigned int s, unsigned int* mt)
{
	mt[0]= s & 0xffffffffUL;
	unsigned int mti;
	for (mti=1; mti<N; mti++) {
		mt[mti] = (1812433253UL * (mt[mti-1] ^ (mt[mti-1] >> 30)) + mti);
		mt[mti] &= 0xffffffffUL;
	}
	mt[N] = mti;
}
__device__ void init_by_seed(unsigned int seed, unsigned int* mt) {
	int i, j, k;
	init_gengrk(19650218UL, mt);
	i=1; j=0;
	k = (N>1 ? N : 1);
	for (; k; k--) {
		mt[i] = (mt[i] ^ ((mt[i-1] ^ (mt[i-1] >> 30)) * 1664525UL)) + seed + j;
		mt[i] &= 0xffffffffUL;
		i++; j++;
		if (i>=N) { mt[0] = mt[N-1]; i=1; }
		if (j>=1) j=0;
	}
	for (k=N-1; k; k--) {
		mt[i] = (mt[i] ^ ((mt[i-1] ^ (mt[i-1] >> 30)) * 1566083941UL)) - i;
		mt[i] &= 0xffffffffUL;
		i++;
		if (i>=N) { mt[0] = mt[N-1]; i=1; }
	}

	mt[0] = 0x80000000UL;
}

__device__ void init_by_array(unsigned int init_key[], unsigned int key_length, unsigned int* mt) {
	int i, j, k;
	init_gengrk(19650218UL, mt);
	i=1; j=0;
	k = (N>key_length ? N : key_length);
	for (; k; k--) {
		mt[i] = (mt[i] ^ ((mt[i-1] ^ (mt[i-1] >> 30)) * 1664525UL)) + init_key[j] + j;
		mt[i] &= 0xffffffffUL;
		i++; j++;
		if (i>=N) { mt[0] = mt[N-1]; i=1; }
		if (j>=key_length) j=0;
	}
	for (k=N-1; k; k--) {
		mt[i] = (mt[i] ^ ((mt[i-1] ^ (mt[i-1] >> 30)) * 1566083941UL)) - i;
		mt[i] &= 0xffffffffUL;
		i++;
		if (i>=N) { mt[0] = mt[N-1]; i=1; }
	}

	mt[0] = 0x80000000UL;
}

__device__ unsigned int gengrk_int32(unsigned int* mt)
{
	unsigned int y;
	if (mt[N] >= N) {
		int kk;

		for (kk=0;kk<N-M;kk++) {
			y = (mt[kk]&UPPER_MASK)|(mt[kk+1]&LOWER_MASK);
			mt[kk] = mt[kk+M] ^ (y >> 1) ^ (y & 0x1UL ? MATRIX_A : 0x0UL);
		}
		for (;kk<N-1;kk++) {
			y = (mt[kk]&UPPER_MASK)|(mt[kk+1]&LOWER_MASK);
			mt[kk] = mt[kk+(M-N)] ^ (y >> 1) ^ (y & 0x1UL ? MATRIX_A : 0x0UL);
		}
		y = (mt[N-1]&UPPER_MASK)|(mt[0]&LOWER_MASK);
		mt[N-1] = mt[M-1] ^ (y >> 1) ^ (y & 0x1UL ? MATRIX_A : 0x0UL);
		mt[N] = 0;
	}

	y = mt[ mt[N] ++ ];

	y ^= (y >> 11);
	y ^= (y << 7) & 0x9d2c5680UL;
	y ^= (y << 15) & 0xefc60000UL;
	y ^= (y >> 18);

	return y;
}

__global__ void init_grk(unsigned int off, unsigned int* buf) {
	unsigned int idx = (blockIdx.x * blockDim.x + threadIdx.x);

	unsigned int seed = (idx + off * NUMCAND) & 0xffffffffU;
	unsigned int ft[N + 1];
	init_by_array(&seed, 1, ft);

	for (int i = 0; i < DIFFLEN; ++i) {
		unsigned int k = 0;
		for (int j = 0; j < 32; ++j) {
			unsigned int a = gengrk_int32(ft);
			gengrk_int32(ft);
			unsigned int u = a > 2147483648;

			if (i*32 + j < ORISIZ)
				k += u << j;
		}
		buf[i + idx * DIFFLEN] = k;
	}
}

__global__ void eval_arr(unsigned int* grk_buf, unsigned int* result, unsigned int* iter) {
	//unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ unsigned int cop[384];

	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	__syncthreads();

	unsigned int cnt = 0;
	for (unsigned int kk = 0; kk < DIFFLEN; kk += 384) {

		cop[threadIdx.x] = grk_buf[threadIdx.x + kk];
		__syncthreads();

		for (unsigned int ii = 0; ii < 384; ++ii) {

			unsigned int n = cop[ii] & iter[  idx + NUMCAND * (ii + kk)];
			//cnt += __popc(n);
			while (n) {
				cnt++;
				n &= (n - 1);
			}
			/*n = n - ((n >> 1) & 0x55555555);
			n = (n & 0x33333333) + ((n >> 2) & 0x33333333);
			cnt += ((n + (n >> 4) & 0xF0F0F0F) * 0x1010101) >> 24;*/
		}
		__syncthreads();
	}
	result[idx] = cnt;
}

int testcuda()
{
	// compare ARRLEN seeds at once
	unsigned int nblocks = NUMCAND / 384;
	unsigned int nthreads = 384;

	unsigned int *grk_buf = NULL;
	unsigned int *device_result = NULL;
	unsigned int *ref = NULL;
	unsigned int *host_result = NULL;

	host_result = (unsigned int*) malloc(NUMCAND * sizeof(unsigned int));
	hipMalloc((void **) &device_result, NUMCAND * sizeof(unsigned int));

	// init grk_buf with n*m values at once
	hipMalloc((void **) &grk_buf, NUMCAND * DIFFLEN *  sizeof(unsigned int));

	dim3 dim_grid(nblocks, 1, 1);
	dim3 dim_block(nthreads, 1, 1);

	hipMalloc((void **) &ref, DEVLEN * sizeof(unsigned int));
	//hipMemcpy(ref, vec,  DEVLEN * sizeof(unsigned int), hipMemcpyHostToDevice);

	unsigned int best = 520;
	unsigned int bestidx = 0;

	//for (unsigned int n = 0; n < 10; ++n;


/*
int deviceCount;
hipGetDeviceCount(&deviceCount);
printf("device count %d\n", deviceCount);

hipDeviceProp_t dP;
hipGetDeviceProperties(&dP, NULL);
//printf("Max threads per block: %d\n", dP.maxThreadsPerBlock);
//printf("Max Threads DIM: %d x %d x %d\n", dP.maxThreadsDim[0], dP.maxThreadsDim[1], dP.maxThreadsDim[2]);
//printf("Max Grid Size: %d x %d x %d\n", dP.maxGridSize[0], dP.maxGridSize[1], dP.maxGridSize[2]);

hipDeviceProp_t* pDeviceProp = &dP;

     printf( "\nDevice Name \t - %s ", pDeviceProp->name );  
     printf( "\n**************************************");  
     printf( "\nTotal Global Memory\t\t -%d KB", pDeviceProp->totalGlobalMem/1024 );  
     printf( "\nShared memory available per block \t - %d KB", pDeviceProp->sharedMemPerBlock/1024 );  
     printf( "\nNumber of registers per thread block \t - %d", pDeviceProp->regsPerBlock );  
     printf( "\nWarp size in threads \t - %d", pDeviceProp->warpSize );  
     printf( "\nMemory Pitch \t - %d bytes", pDeviceProp->memPitch );  
     printf( "\nMaximum threads per block \t - %d", pDeviceProp->maxThreadsPerBlock );  
     printf( "\nMaximum Thread Dimension (block) \t - %d %d %d", pDeviceProp->maxThreadsDim[0], pDeviceProp->maxThreadsDim[1], pDeviceProp->maxThreadsDim[2] );  
     printf( "\nMaximum Thread Dimension (grid) \t - %d %d %d", pDeviceProp->maxGridSize[0], pDeviceProp->maxGridSize[1], pDeviceProp->maxGridSize[2] );  
     printf( "\nTotal constant memory \t - %d bytes", pDeviceProp->totalConstMem );  
     printf( "\nCUDA ver \t - %d.%d", pDeviceProp->major, pDeviceProp->minor );  
     printf( "\nClock rate \t - %d KHz", pDeviceProp->clockRate );  
     printf( "\nTexture Alignment \t - %d bytes", pDeviceProp->textureAlignment );  
     printf( "\nDevice Overlap \t - %s", pDeviceProp-> deviceOverlap?"Allowed":"Not Allowed" );  
     printf( "\nNumber of Multi processors \t - %d\n", pDeviceProp->multiProcessorCount );  

*/

/*
	for (unsigned int n = 0; n < 30000; ++n) {
	//for (unsigned int n = 50; n < 100; ++n) {

		if (n % 10 == 0) { printf("# iter %d\n", n); fflush(stdout); }

		init_grk <<< dim_grid, dim_block >>> (n, grk_buf);
		for (unsigned int i = 0; i < NUMCAND; ++i) {
			eval_arr <<< dim_grid, dim_block >>> (grk_buf + i * DIFFLEN, device_result, ref);
			hipMemcpy(host_result, device_result, NUMCAND * sizeof(unsigned int), hipMemcpyDeviceToHost);

			unsigned int s1 = 0;
			unsigned int s2 = 0;
			for (unsigned int k = 0; k < NUMCAND; ++k) {
				if (host_result[k] > s1) s1 = host_result[k];
				else if (valz[k] - host_result[k] > s2) s2 = valz[k] - host_result[k];
			}
			if (s1 + s2 >= 530) {
				if ((s1 + s2) > best) {
					best = s1 + s2;
					bestidx = i + n * NUMCAND;
				}
				unsigned int ss1 = 0;
				unsigned int ss2 = 0;
				unsigned int sss1 = 0;
				unsigned int sss2 = 0;
				for (unsigned int k = 0; k < NUMCAND; ++k) {
					if      (host_result[k] >= s1          ) { s1 = host_result[k];           ss1 = seedz[k]; sss1 = seedz_off[k]; }
					else if (valz[k] - host_result[k] >= s2) { s2 = valz[k] - host_result[k]; ss2 = seedz[k]; sss2 = seedz_off[k]; }
				}
				printf("(%d, %d, %d, %d, %d, %d), # %d+%d  best %d, %d\n", i + n * NUMCAND, ss1, sss1, ss2, sss2, s1+s2, s1, s2, best, bestidx);
				fflush(stdout);
			}
		}

	}

	hipFree(grk_buf);
	hipFree(device_result);
	hipFree(ref);
	free(host_result);
*/

	return 0;
}

